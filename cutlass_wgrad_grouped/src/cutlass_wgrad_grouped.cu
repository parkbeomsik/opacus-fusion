#include <vector>
// #include "base_operation.h"
// #include "cutlass_error.h"
#include "cutlass_wgrad_grouped.h"
#include "initialize_all.h"
#include "wgrad_grouped_operation.h"

#include "hip/hip_runtime.h"
#include "cuda_error_helper.h"

#include "cutlass/conv/conv2d_problem_size.h"
#include "cutlass/layout/tensor.h"


namespace cutlass_wgrad_grouped {

void * _device_problems = NULL;

std::vector<Operation *> operations;

std::vector<void *> device_workspaces;
// std::vector<void *> host_workspaces;

std::vector<OperationWithWorkspace> operations_with_workspaces;

void ** device_ptr_A;
void ** device_ptr_B;
void ** device_ptr_C;
void ** device_ptr_D;

int problem_count;


void initialize() {

    initialize_swgrad_grouped(operations);
}

void initialize_problems(std::vector<Conv2dConfig> const & host_configs) {

    using namespace cutlass::conv;
    
    problem_count = host_configs.size();
    // printf("problem count = %d\n", problem_count);

    // Set problem sizes in host memory first
    std::vector<Conv2dProblemSize> host_problems;
    for (int i = 0; i < problem_count; ++i) {
        Conv2dConfig host_config = host_configs.at(i);

        // Set single problem in host
        Conv2dProblemSize problem(host_config.N, 
                                  host_config.H, host_config.W, host_config.C,
                                  host_config.K, host_config.R, host_config.S,
                                  host_config.P, host_config.Q,
                                  host_config.pad_h, host_config.pad_w,
                                  host_config.stride_h, host_config.stride_w,
                                  host_config.dilation_h, host_config.dilation_w,
                                  Mode::kCrossCorrelation, host_config.split_k_slices);

        host_problems.push_back(problem);
    }

    assert(host_problems.size() == problem_count);

    // Set problems in device memory
    if (_device_problems != NULL) {
        checkCudaErrors(hipFree(_device_problems));
    }
    checkCudaErrors(hipMalloc(&_device_problems, 
                                (size_t)sizeof(Conv2dProblemSize)*problem_count));
    checkCudaErrors(hipMemcpy((void *)_device_problems, (void *)host_problems.data(), 
                               (size_t)sizeof(Conv2dProblemSize)*problem_count,
                               hipMemcpyHostToDevice));

    // Set problems of operations
    using TensorRefAll = cutlass::TensorRef<float, cutlass::layout::TensorNHWC>;
    std::vector<TensorRefAll> host_ref_A;
    std::vector<TensorRefAll> host_ref_B;
    std::vector<TensorRefAll> host_ref_C;
    std::vector<TensorRefAll> host_ref_D;

    TensorRefAll *device_ref_A;
    TensorRefAll *device_ref_B;
    TensorRefAll *device_ref_C;
    TensorRefAll *device_ref_D;
    checkCudaErrors(hipMalloc(&device_ref_A, sizeof(TensorRefAll)*problem_count));
    checkCudaErrors(hipMalloc(&device_ref_B, sizeof(TensorRefAll)*problem_count));
    checkCudaErrors(hipMalloc(&device_ref_C, sizeof(TensorRefAll)*problem_count));
    checkCudaErrors(hipMalloc(&device_ref_D, sizeof(TensorRefAll)*problem_count));
    for (auto device_workspace : device_workspaces){
        checkCudaErrors(hipFree(device_workspace));
    }
    device_workspaces.clear();
    device_workspaces.push_back(device_ref_A);
    device_workspaces.push_back(device_ref_B);
    device_workspaces.push_back(device_ref_C);
    device_workspaces.push_back(device_ref_D);

    wGradGroupedConfig wgrad_config = {(Conv2dProblemSize *)_device_problems,
                                       problem_count,
                                       (void *)device_ref_A,
                                       (void *)device_ref_B,
                                       (void *)device_ref_C,
                                       (void *)device_ref_D,
                                       &host_problems[0]};
    for (auto operation : operations) {
        void * host_workspace = malloc(operation->get_host_workspace_size());
        operations_with_workspaces.push_back(OperationWithWorkspace({operation, host_workspace}));
    }

    size_t workspace_size = operations_with_workspaces.at(0).operation->get_workspace_size(&wgrad_config, operations_with_workspaces.at(0).host_workspace);

    void * device_semaphore;
    checkCudaErrors(hipMalloc(&device_semaphore, workspace_size));
    device_workspaces.push_back(device_semaphore);

    for (auto operation_with_workspace : operations_with_workspaces) {
        checkCUTLASS(operation_with_workspace.operation->initialize(&wgrad_config, device_semaphore, operation_with_workspace.host_workspace));
    }

    assert(operations_with_workspaces.size() == operations.size());
}

void finalize() {
    for (auto device_workspace : device_workspaces){
        checkCudaErrors(hipFree(device_workspace));
    }
    for (auto operation_with_workspace : operations_with_workspaces) {
        free(operation_with_workspace.host_workspace);
        free(operation_with_workspace.operation);
    }
    
    device_workspaces.clear();
    operations_with_workspaces.clear();
    operations.clear();

    checkCudaErrors(hipFree(_device_problems));
    _device_problems = NULL;
}

OperationWithWorkspace get_best_operation(void ** ptr_A,
                              void ** ptr_B,
                              void ** ptr_C,
                              void ** ptr_D) {

    assert(operations_with_workspaces.size() == operations.size());

    std::vector<float> runtime_ms_list(operations.size(), 100000.0);
    // runtime_ms_list.resize(operations.size());

    for (int i = 0; i < operations.size(); ++i) {
        auto operation = operations.at(i);
        auto host_workspace = operations_with_workspaces.at(i).host_workspace;

        checkCUTLASS(operation->update_ptrs(ptr_A, ptr_B, ptr_C, ptr_D, problem_count, host_workspace));

        hipEvent_t events[2];

        // Warm up
        for (auto & event : events) {
            checkCudaErrors(hipEventCreate(&event));
        }

        // Record an event at the start of a series of GEMM operations
        checkCudaErrors(hipEventRecord(events[0]));

        Status result;
        for (int iter = 0; iter < 3; ++iter) {
            result = operation->run(host_workspace);
        }

        checkCudaErrors(hipEventRecord(events[1]));
        checkCudaErrors(hipEventSynchronize(events[1]));

        float runtime_ms;
        checkCudaErrors(hipEventElapsedTime(&runtime_ms, events[0], events[1]));

        // checkCudaErrors(hipDeviceSynchronize());

        if (result == Status::kSuccess && runtime_ms < 30.0) {
            // Measure runtime

            // Record an event at the start of a series of GEMM operations
            checkCudaErrors(hipEventRecord(events[0]));

            for (int iter = 0; iter < 20; ++iter) {
                result = operation->run(host_workspace);
            }

            checkCudaErrors(hipEventRecord(events[1]));
            checkCudaErrors(hipEventSynchronize(events[1]));

            checkCudaErrors(hipEventElapsedTime(&runtime_ms, events[0], events[1]));

            for (auto & event : events) {
                checkCudaErrors(hipEventDestroy(event));
            }
        }
        else {
            runtime_ms = 2000.0;
        }

        if (result == Status::kSuccess) {
            // If CUTLASS is so slow for this problem, it stops early.
            printf("runtime_ms = %f\t%s\n", runtime_ms, operation->name.c_str());
            if (runtime_ms > 3000.0) {
                break;
            }
            runtime_ms_list.at(i) = runtime_ms;
        } else {
            // Means it failed
            runtime_ms_list.at(i) = 100000.0;
        }
        // 
    }

    assert(runtime_ms_list.size() == operations.size());

    float min_runtime_ms = 10000.0;
    OperationWithWorkspace best_operation {NULL, NULL};
    for (int i = 0; i < runtime_ms_list.size(); ++i) {
        if (runtime_ms_list.at(i) < min_runtime_ms) {
            min_runtime_ms = runtime_ms_list.at(i);
            best_operation = operations_with_workspaces.at(i);
        }
    }

    return best_operation;
}

Status run(OperationWithWorkspace operation_with_workspace) {
    return operation_with_workspace.operation->run(operation_with_workspace.host_workspace);
}

Status update_ptrs(OperationWithWorkspace operation_with_workspace,
                   void ** ptr_A,
                   void ** ptr_B,
                   void ** ptr_C,
                   void ** ptr_D,
                   int problem_count) {
    return operation_with_workspace.operation->update_ptrs(ptr_A, ptr_B, ptr_C, ptr_D, problem_count, operation_with_workspace.host_workspace);
}


} // namespace cutlass_wgrad_grouped