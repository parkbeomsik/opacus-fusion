#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include "ATen/cudnn/Handles.h"
#include <c10/cuda/CUDAStream.h>
#include <c10/cuda/CUDACachingAllocator.h>

#include <vector>
#include <chrono>
#include <algorithm>
#include <cmath>
#include <random>

#include "structure.h"
#include "error_helper.h"
#include "utils.h"

#include <hiprand/hiprand_kernel.h>

bool _quantize_first_run = true;
hiprandState *d_state;

std::mt19937 gen(1234);
std::uniform_int_distribution<unsigned long long> dis(0, ULLONG_MAX);

hipblasHandle_t quant_handle;
hipStream_t quant_stream;
// c10::cuda::CUDAStream torch_quant_stream;

torch::Tensor workspace_tensor;

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void quantize_int8_cuda_kernel(
    float* m,
    float* max_m,
    int8_t* q,
    int num_elem,
    float * scale,
    float * workspace,
    unsigned long long seed) {
  // Compute max of abs
  __shared__ float s_float[1024];
  int c = blockIdx.x * blockDim.x + threadIdx.x;
  float local_max = 0;
  while (c < num_elem){
    local_max = max(abs(m[c]), local_max);
    c += blockDim.x * gridDim.x;
  }
  s_float[threadIdx.x] = local_max;
  int num_threads_to_reduce = 512;
  while (num_threads_to_reduce > 0) {
    if (c < num_threads_to_reduce) {
      s_float[threadIdx.x] = max(s_float[threadIdx.x], s_float[threadIdx.x + num_threads_to_reduce]);
    }
    num_threads_to_reduce /= 2;
    __syncthreads();
  }
  if (threadIdx.x == 0) {
    workspace[blockIdx.x] = s_float[0];
  }
  __syncwarp();
  local_max = 0;
  if (blockIdx.x == 0) {
    while (c < gridDim.x){
      local_max = max(workspace[c], local_max);
      c += blockDim.x;
    }
    s_float[threadIdx.x] = 0;
    s_float[threadIdx.x] = local_max;
    __syncthreads();
    num_threads_to_reduce = 512;
    while (num_threads_to_reduce > 0) {
      if (c < num_threads_to_reduce) {
        s_float[threadIdx.x] = max(s_float[threadIdx.x], s_float[threadIdx.x + num_threads_to_reduce]);
      }
      num_threads_to_reduce /= 2;
      __syncthreads();
    }
    if (threadIdx.x == 0) {
      *max_m = s_float[0];
    }
  }

  __syncwarp();

  // Quantize
  // column index
  c = blockIdx.x * blockDim.x + threadIdx.x;
  float max_m_ = *max_m;
  while (c < num_elem){
    hiprandState s;
    hiprand_init(seed + c, 0, 0, &s);
    if (c == 0) {
      (*scale) = (1 << 6)/(max_m_);
    }
    auto elem = m[c] * (1 << 6)/(max_m_);

    q[c] = (int8_t)((elem - (int32_t)elem) < hiprand_uniform(&s) ? (int32_t)elem : elem + 1);

    c += blockDim.x * gridDim.x;
  }
}

std::vector<std::vector<torch::Tensor>> quantize_int8(std::vector<torch::Tensor>& m_list) {
  if (_quantize_first_run) {
    checkCudaErrors(hipStreamCreate(&quant_stream));
    checkCUBLAS(hipblasCreate(&quant_handle));
    checkCUBLAS(hipblasSetStream(quant_handle ,quant_stream));
    checkCUBLAS(hipblasSetPointerMode(quant_handle, HIPBLAS_POINTER_MODE_DEVICE));

    _quantize_first_run = false;

    workspace_tensor = torch::empty({100000}, torch::TensorOptions().device(torch::kCUDA, 0));

    // torch_quant_stream = c10::cuda::getStreamFromExternal(quant_stream, 0);
  }

  c10::cuda::setCurrentCUDAStream(c10::cuda::getStreamFromExternal(quant_stream, 0));

  std::vector<std::vector<torch::Tensor>> q_list;

  for(auto& m : m_list) {
    // auto abs = torch::abs(m);
    auto max_m_idx = torch::empty({1}, torch::TensorOptions().device(torch::kCUDA, 0).dtype(torch::kInt));
    // checkCUBLAS(hipblasIsamax(quant_handle, abs.numel(), (float *)abs.data_ptr(), 1, (int *)max_m_idx.data_ptr()));
    // auto max_m = torch::abs(m).max();
    // auto scale = (1 << 6)/max_m;
    // auto scale = (1 << 6)/m.flatten().index({max_m_idx});

    // auto m_shape = m.sizes();
    // m = m.reshape({m.sizes()[0], -1});
    auto batch_size = m.sizes()[0];
    auto num_params = m.numel() / batch_size;
    auto num_elem = m.numel();

    const int threads = 1024;
    const dim3 blocks(std::min((num_elem + threads - 1) / threads, (int64_t)360), 1);

    auto q = torch::empty(m.sizes(), torch::TensorOptions().device(torch::kCUDA, 0).dtype(torch::kInt8));
    auto scale = torch::empty({1}, torch::TensorOptions().device(torch::kCUDA, 0));

    unsigned long long seed = dis(gen);

    quantize_int8_cuda_kernel<<<blocks, threads, 0, quant_stream>>>(
        (float *) m.data_ptr(),
        (float *)max_m_idx.data_ptr(),
        (int8_t *) q.data_ptr(),
        (int) num_elem,
        (float *)scale.data_ptr(),
        (float *)workspace_tensor.data_ptr(),
        seed);

    q_list.push_back({q, scale});
  }

  c10::cuda::setCurrentCUDAStream(c10::cuda::getDefaultCUDAStream());

  return q_list;
}

__global__ void int32_to_float32_cuda_kernel(
    int32_t *m,
    float *out,
    int n) {
  // column index
  const int c = blockIdx.x * blockDim.x + threadIdx.x;
  if (c < n){
    out[c] = (float)m[c];
  }
}

void int32_to_float32(torch::Tensor m, torch::Tensor out, int n) {

  const int threads = 1024;
  const dim3 blocks((n + threads - 1) / threads, 1);

  int32_to_float32_cuda_kernel<<<blocks, threads>>>(
      (int32_t *)m.data_ptr(),
      (float *)out.data_ptr(),
      n);

  return;
}