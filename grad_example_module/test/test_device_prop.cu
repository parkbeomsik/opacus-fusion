#include "hip/hip_runtime.h"
#include <iostream>

int main(void) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    printf("%d.%d\n", deviceProp.major, deviceProp.minor);
}