
#include <vector>
#include <iostream>
#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/conv/kernel/default_conv2d_wgrad.h"
#include "cutlass/conv/device/implicit_gemm_convolution.h"

#include "cutlass/reduction/device/reduce_split_k.h"
#include "cutlass/reduction/thread/reduction_operators.h"

#include "hip/hip_runtime.h"

#include "error_helper.h"

// The code section below describes datatype for input, output tensors and computation between
// elements
// In Wgrad, fp32 accumulation is necessary in practice.
using ElementAccumulator = int32_t;                  // Data type of accumulator
using ElementComputeEpilogue = float;              // Data type of epilogue computation (alpha, beta)
using ElementInputA = int8_t;             // Data type of elements in input tensor
using ElementInputB = int8_t;             // Data type of elements in input tensor
using ElementOutput = float;                       // Data type of elements in output tensor
using ElementC = ElementOutput;
using ElementCompute = ElementComputeEpilogue;
using LayoutInputA = cutlass::layout::TensorNHWC;
using LayoutInputB = cutlass::layout::TensorNHWC;
using LayoutOutput = cutlass::layout::TensorNHWC;

// This code section describes whether you want to use tensor cores or regular SIMT cores on GPU SM
using MMAOp = cutlass::arch::OpClassSimt;

// This code section describes CUDA SM architecture number
using SmArch = cutlass::arch::Sm75;

// This code section describes the tile size a thread block will compute
using ThreadblockShape = cutlass::gemm::GemmShape<128, 256, 8>; // Threadblock tile shape

// This code section describes tile size a warp will compute
using WarpShape = cutlass::gemm::GemmShape<16, 16, 8>;          // Warp tile shape

// This code section describes the size of MMA op
using InstructionShape = cutlass::gemm::GemmShape<1, 1, 4>;    // TensorCore instruction shape

// This code section describes how threadblocks are scheduled on GPU
using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;

// Number of pipelines you want to use
constexpr int NumStages = 3;

// This code section describe iterator algorithm selected is Analytic or Optimized
static cutlass::conv::IteratorAlgorithm const IteratorAlgorithm = cutlass::conv::IteratorAlgorithm::kOptimized;

// We need two epilogue functors - one for GEMM and another for the final reduction.
// The epilogue for GEMM is not used, but needed to instantiate the CUTLASS kernel template.
// Note that, when the input is fp16 and accumulation is fp32, the output of GEMM needs to be fp32,
// the final reduction is done in fp32, and the reduction epilogue converts fp32 outputs to fp16.
// Therefore, the output type of the GEMM epilogue is ElementCompute, not ElementOutput.

// This code section describes the epilogue part of the kernel, we use default value
using EpilogueOpGEMM = cutlass::epilogue::thread::LinearCombination<
    ElementCompute,                                     // Data type of output matrix.
    1,  // The number of elements per vectorized.
    // memory access. This becomes the vector width of
    // math instructions in the epilogue too.
    ElementAccumulator,                                // Data type of accumulator
    ElementComputeEpilogue>;                           // Data type for alpha/beta in linear combination

// The epilogue functor for reduction. This is the one that is actually used.
using EpilogueOpReduction = cutlass::epilogue::thread::LinearCombination<
    ElementOutput,                                     // Data type of output matrix.
    1,  // The number of elements per vectorized.
    // memory access. This becomes the vector width of
    // math instructions in the epilogue too.
    ElementAccumulator,                                // Data type of accumulator
    ElementComputeEpilogue>;                           // Data type for alpha/beta in lin

using Conv2dWgradKernel = typename cutlass::conv::kernel::DefaultConv2dWgrad<
    ElementInputA, LayoutInputA,
    ElementInputB, LayoutInputB,
    ElementAccumulator, LayoutOutput,
    ElementAccumulator,
    MMAOp,
    SmArch,
    ThreadblockShape,
    WarpShape,
    InstructionShape,
    EpilogueOpGEMM,
    SwizzleThreadBlock,
    NumStages,
    cutlass::arch::OpMultiplyAdd,
    IteratorAlgorithm
    >::Kernel;

using ImplicitGemm = cutlass::conv::device::ImplicitGemmConvolution<Conv2dWgradKernel>;

using EpilogueOutputOp = EpilogueOpReduction;

/// Reduction kernel
using ReductionOp = cutlass::reduction::thread::ReduceAdd<
    ElementAccumulator,
    typename EpilogueOutputOp::ElementAccumulator,
    EpilogueOutputOp::kCount
  >;

using ReductionKernel = cutlass::reduction::kernel::ReduceSplitK<
    cutlass::MatrixShape<4, 32 * EpilogueOutputOp::kCount>,
    EpilogueOutputOp,
    ReductionOp
  >;

using ReductionDevice = cutlass::reduction::device::ReduceSplitK<ReductionKernel>;
using ReductionStrideIndex = typename ReductionDevice::StrideIndex;

hipError_t cutlass_simt_iwgrad(
  int8_t * ograd,
  int8_t * actv,
  float * wgrad,
  void * workspace,
  int N,
  int H,
  int W,
  int C,
  int K,
  int R,
  int S,
  int P,
  int Q,
  int pad_h,
  int pad_w,
  int stride_h,
  int stride_w,
  int dilation_h,
  int dilation_w,
  int split_k_slices,
  float alpha,
  float beta,
  hipStream_t stream) {

  ImplicitGemm implicit_gemm;

  cutlass::conv::Mode mode = cutlass::conv::Mode::kCrossCorrelation;

  cutlass::conv::Conv2dProblemSize problem_size(
    N,
    H,
    W,
    C,
    K,
    R,
    S,
    P,
    Q,
    pad_h,
    pad_w,
    stride_h,
    stride_w,
    dilation_h,
    dilation_w,
    mode,
    split_k_slices
  );

  using cutlass::layout::TensorNHWC;

  cutlass::conv::SplitKMode const split_k_mode = cutlass::conv::SplitKMode::kParallel;

  cutlass::TensorRef<ElementInputA, LayoutInputA> tensor_a(ograd, TensorNHWC::packed(cutlass::Tensor4DCoord(N, P, Q, K)));
  cutlass::TensorRef<ElementInputB, LayoutInputB> tensor_b(actv, TensorNHWC::packed(cutlass::Tensor4DCoord(N, H, W, C)));
  cutlass::TensorRef<ElementOutput, LayoutOutput> tensor_c(wgrad, TensorNHWC::packed(cutlass::Tensor4DCoord(K, R, S, C)));
  cutlass::TensorRef<ElementOutput, LayoutOutput> tensor_d(wgrad, TensorNHWC::packed(cutlass::Tensor4DCoord(K, R, S, C)));

  typename ImplicitGemm::Arguments arguments{
    problem_size,
    tensor_a,
    tensor_b,
    {nullptr, TensorNHWC()},
    {nullptr, TensorNHWC()},
    {ElementCompute(alpha), ElementCompute(beta)},
    split_k_mode
  };

  checkCutlassRaw(implicit_gemm.initialize(arguments, workspace));
  checkCutlassRaw(implicit_gemm(stream));

  // Do reduction
  ReductionDevice reduction_op;
  static cutlass::conv::Operator const kConvolutionalOperator = ImplicitGemm::kConvolutionalOperator;
  typename ReductionDevice::Arguments reduction_args(
      cutlass::conv::implicit_gemm_problem_size(kConvolutionalOperator, problem_size).mn(),
      problem_size.split_k_slices,
      cutlass::conv::implicit_gemm_tensor_c_size(kConvolutionalOperator, problem_size),
      // Reduction input
      {
          reinterpret_cast<ElementAccumulator*> (workspace),
          ReductionStrideIndex(tensor_c.stride()[ImplicitGemm::ImplicitGemmKernel::kTensorCStrideIdx])
      },
      // Destination
      {
          tensor_d.data(),
          ReductionStrideIndex(tensor_d.stride()[ImplicitGemm::ImplicitGemmKernel::kTensorCStrideIdx])
      },
      // Source
      {
          tensor_c.data(),
          ReductionStrideIndex(tensor_c.stride()[ImplicitGemm::ImplicitGemmKernel::kTensorCStrideIdx])
      },
      {alpha, beta}
  );

  checkCutlassRaw(reduction_op.initialize(reduction_args, nullptr));
  checkCutlassRaw(reduction_op());

  return hipSuccess;
  }

size_t cutlass_simt_iwgrad_get_workspace(
  int N,
  int H,
  int W,
  int C,
  int K,
  int R,
  int S,
  int P,
  int Q,
  int pad_h,
  int pad_w,
  int stride_h,
  int stride_w,
  int dilation_h,
  int dilation_w,
  int split_k_slices) {  

  ImplicitGemm implicit_gemm;

  cutlass::conv::Mode mode = cutlass::conv::Mode::kCrossCorrelation;

  cutlass::conv::Conv2dProblemSize problem_size(
    N,
    H,
    W,
    C,
    K,
    R,
    S,
    P,
    Q,
    pad_h,
    pad_w,
    stride_h,
    stride_w,
    dilation_h,
    dilation_w,
    mode,
    split_k_slices
  );

  using cutlass::layout::TensorNHWC;

  cutlass::conv::SplitKMode const split_k_mode = cutlass::conv::SplitKMode::kParallel;

  cutlass::TensorRef<ElementInputA, LayoutInputA> tensor_a(NULL, TensorNHWC::packed(cutlass::Tensor4DCoord(N, P, Q, K)));
  cutlass::TensorRef<ElementInputB, LayoutInputB> tensor_b(NULL, TensorNHWC::packed(cutlass::Tensor4DCoord(N, H, W, C)));

  typename ImplicitGemm::Arguments arguments{
    problem_size,
    tensor_a,
    tensor_b,
    {nullptr, TensorNHWC()},
    {nullptr, TensorNHWC()},
    {ElementCompute(1.0), ElementCompute(0.0)},
    split_k_mode
  };

  size_t workspace_size = implicit_gemm.get_workspace_size(arguments);

  return workspace_size;
}

int main(int argc, char * argv[]) {
  int N = atoi(argv[1]);
  int H = atoi(argv[2]);
  int W = atoi(argv[3]);
  int C = atoi(argv[4]);
  int K = atoi(argv[5]);
  int R = atoi(argv[6]);
  int S = atoi(argv[7]);
  int P = atoi(argv[8]);
  int Q = atoi(argv[9]);
  int pad_h = atoi(argv[10]);
  int pad_w = atoi(argv[11]);
  int stride_h = atoi(argv[12]);
  int stride_w = atoi(argv[13]);
  int dilation_h = atoi(argv[14]);
  int dilation_w = atoi(argv[15]);
  int split_k_slices = atoi(argv[16]);

  size_t ws_size = cutlass_simt_iwgrad_get_workspace(N,
                                                     H,
                                                     W,
                                                     C,
                                                     K,
                                                     R,
                                                     S,
                                                     P,
                                                     Q,
                                                     pad_h,
                                                     pad_w,
                                                     stride_h,
                                                     stride_w,
                                                     dilation_h,
                                                     dilation_w,
                                                     split_k_slices);

  void * ograd;
  void * actv;
  void * wgrad;
  void * ws;
  checkCudaErrors(hipMalloc(&ograd, sizeof(int8_t)*N*K*P*Q));
  checkCudaErrors(hipMalloc(&actv, sizeof(int8_t)*N*C*H*W));
  checkCudaErrors(hipMalloc(&wgrad, sizeof(float)*K*C*R*S));
  checkCudaErrors(hipMalloc(&ws, ws_size));

  // Warm up
  for (int i = 0; i < 3; ++i) {
    cutlass_simt_iwgrad((int8_t *)ograd,
                        (int8_t *)actv,
                        (float *)wgrad,
                        ws,
                        N,
                        H,
                        W,
                        C,
                        K,
                        R,
                        S,
                        P,
                        Q,
                        pad_h,
                        pad_w,
                        stride_h,
                        stride_w,
                        dilation_h,
                        dilation_w,
                        split_k_slices,
                        1.0,
                        0.0,
                        NULL);
  }

  hipEvent_t events[2];
  checkCudaErrors(hipEventCreate(&events[0]));
  checkCudaErrors(hipEventCreate(&events[1]));

  checkCudaErrors(hipEventRecord(events[0]));

  // Measure runtime_ms
  for (int i = 0; i < 20; ++i) {
    cutlass_simt_iwgrad((int8_t *)ograd,
                        (int8_t *)actv,
                        (float *)wgrad,
                        ws,
                        N,
                        H,
                        W,
                        C,
                        K,
                        R,
                        S,
                        P,
                        Q,
                        pad_h,
                        pad_w,
                        stride_h,
                        stride_w,
                        dilation_h,
                        dilation_w,
                        split_k_slices,
                        1.0,
                        0.0,
                        NULL);
  }

  checkCudaErrors(hipEventRecord(events[1]));
  checkCudaErrors(hipDeviceSynchronize());

  float runtime_ms = 0.0;
  checkCudaErrors(hipEventElapsedTime(&runtime_ms, events[0], events[1]));

  std::cout << runtime_ms / 20.0 << std::endl;
}

