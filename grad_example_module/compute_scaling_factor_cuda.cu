#include <hip/hip_runtime.h>

__global__ void compute_scaling_factor_cuda_kernel(float *out, 
                                                   const float *norm,
                                                   float max_norm) {
  *(out + threadIdx.x) = min((max_norm / (*(norm + threadIdx.x) + 1e-6)), 1.0);
}

void compute_scaling_factor_cuda(float *out, 
                                const float *norm,
                                float max_norm,
                                int num_rows_to_compute) {

    compute_scaling_factor_cuda_kernel<<<1, num_rows_to_compute>>>(out, norm, max_norm);
}

__global__ void compute_scaling_factor2_cuda_kernel(float *out, 
                                                   const float *norm,
                                                   const float *norm2,
                                                   float max_norm) {
  *out = min((max_norm / (sqrt(powf(*norm, 2) + powf(*norm2, 2)) + 1e-6)), 1.0);
}

void compute_scaling_factor2_cuda(float *out, 
                                const float *norm,
                                const float *norm2,
                                float max_norm) {

    compute_scaling_factor2_cuda_kernel<<<1, 1>>>(out, norm, norm2, max_norm);
}